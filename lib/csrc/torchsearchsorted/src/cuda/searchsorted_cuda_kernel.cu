#include "hip/hip_runtime.h"
#include "searchsorted_cuda_kernel.h"

template <typename scalar_t>
__device__
int eval(scalar_t val, scalar_t *a, int64_t row, int64_t col, int64_t ncol, bool side_left)
{
    /* Evaluates whether a[row,col] < val <= a[row, col+1]*/

    if (col == ncol - 1)
    {
      // special case: we are on the right border
      if (a[row * ncol + col] <= val){
        return 1;}
      else {
        return -1;}
    }
    bool is_lower;
    bool is_next_higher;

    if (side_left) {
      // a[row, col] < v <= a[row, col+1]
      is_lower = (a[row * ncol + col] < val);
      is_next_higher = (a[row*ncol + col + 1] >= val);
    } else {
      // a[row, col] <= v < a[row, col+1]
      is_lower = (a[row * ncol + col] <= val);
      is_next_higher = (a[row * ncol + col + 1] > val);
    }
    if (is_lower && is_next_higher) {
        // we found the right spot
        return 0;
    } else if (is_lower) {
    	// answer is on the right side
        return 1;
    } else {
    	// answer is on the left side
        return -1;
    }
}

template <typename scalar_t>
__device__
int binary_search(scalar_t *a, int64_t row, scalar_t val, int64_t ncol, bool side_left)
{
  /* Look for the value `val` within row `row` of matrix `a`, which
  has `ncol` columns.

  the `a` matrix is assumed sorted in increasing order, row-wise

  Returns 
  * -1 if `val` is smaller than the smallest value found within that row of `a`
  * `ncol` - 1 if `val` is larger than the largest element of that row of `a`
  * Otherwise, return the column index `res` such that:
    - a[row, col] < val <= a[row, col+1]. (if side_left), or 
    - a[row, col] < val <= a[row, col+1] (if not side_left).
   */

  //start with left at 0 and right at number of columns of a
  int64_t right = ncol;
  int64_t left = 0;

  while (right >= left) {
      // take the midpoint of current left and right cursors
      int64_t mid = left + (right-left)/2;

      // check the relative position of val: are we good here ?
      int rel_pos = eval(val, a, row, mid, ncol, side_left);
      // we found the point
      if(rel_pos == 0) {
          return mid;
      } else if (rel_pos > 0) {
        if (mid==ncol-1){return ncol-1;}
        // the answer is on the right side
        left = mid;
      } else {
        if (mid==0){return -1;}
        right = mid;
      }
  }
  return -1;
}

template <typename scalar_t>
__global__
void searchsorted_kernel(
  int64_t *res,
  scalar_t *a,
  scalar_t *v,
  int64_t nrow_res, int64_t nrow_a, int64_t nrow_v, int64_t ncol_a, int64_t ncol_v, bool side_left)
{
    // get current row and column
    int64_t row = blockIdx.y*blockDim.y+threadIdx.y;
    int64_t col = blockIdx.x*blockDim.x+threadIdx.x;

    // check whether we are outside the bounds of what needs be computed.
    if ((row >= nrow_res) || (col >= ncol_v)) {
      return;}

    // get the value to look for
    int64_t row_in_v = (nrow_v==1) ? 0: row;
    int64_t row_in_a = (nrow_a==1) ? 0: row;
    int64_t idx_in_v = row_in_v*ncol_v+col;
    int64_t idx_in_res = row*ncol_v+col;

    // apply binary search
    res[idx_in_res] = binary_search(a, row_in_a, v[idx_in_v], ncol_a, side_left)+1;
}


void searchsorted_cuda(
  at::Tensor a,
  at::Tensor v,
  at::Tensor res,
  bool side_left){

      // Get the dimensions
      auto nrow_a = a.size(/*dim=*/0);
      auto nrow_v = v.size(/*dim=*/0);
      auto ncol_a = a.size(/*dim=*/1);
      auto ncol_v = v.size(/*dim=*/1);

      auto nrow_res = fmax(double(nrow_a), double(nrow_v));

      // prepare the kernel configuration
      dim3 threads(ncol_v, nrow_res);
      dim3 blocks(1, 1);
      if (nrow_res*ncol_v > 1024){
         threads.x = int(fmin(double(1024), double(ncol_v)));
         threads.y = floor(1024/threads.x);
         blocks.x = ceil(double(ncol_v)/double(threads.x));
         blocks.y = ceil(double(nrow_res)/double(threads.y));
      }

      AT_DISPATCH_ALL_TYPES(a.type(), "searchsorted cuda", ([&] {
        searchsorted_kernel<scalar_t><<<blocks, threads>>>(
          res.data<int64_t>(),
          a.data<scalar_t>(),
          v.data<scalar_t>(),
          nrow_res, nrow_a, nrow_v, ncol_a, ncol_v, side_left);
      }));

  }
